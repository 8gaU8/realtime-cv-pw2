#include "hip/hip_runtime.h"
// === standard libraries ===
#include <iostream>
#include <chrono> // for high_resolution_clock
// #include <cfloat>

// === OpenCV with CUDA ===
#include <opencv2/opencv.hpp>
#include <opencv2/cudawarping.hpp>
#include <opencv2/core/cuda/common.hpp>
#include <opencv2/core/cuda/border_interpolate.hpp>
#include <opencv2/core/cuda/vec_traits.hpp>
#include <opencv2/core/cuda/vec_math.hpp>

#include "hip/hip_vector_types.h"
#include "anaglyphMethods.cuh"

using namespace std;

void makeGaussianKernel(int kernelSizeDiv2, float sigma, cv::Mat_<float> &kernelMat)
{
    for (int i = 0; i < 2 * kernelSizeDiv2 + 1; i++)
    {
        for (int j = 0; j < 2 * kernelSizeDiv2 + 1; j++)
        {
            float x = i - kernelSizeDiv2;
            float y = j - kernelSizeDiv2;
            kernelMat(i, j) = exp(-(x * x + y * y) / (2 * sigma * sigma)) / (2 * M_PI * sigma * sigma);
        }
    }
}

__global__ void applyGaussianFilter(
    const cv::cuda::PtrStep<uchar3> src,
    cv::cuda::PtrStep<uchar3> dst,
    int rows,
    int cols,
    int kernelSizeDiv2,
    const cv::cuda::PtrStep<float> kernelMat)

{
    const int dst_x = blockDim.x * blockIdx.x + threadIdx.x;
    const int dst_y = blockDim.y * blockIdx.y + threadIdx.y;
    int minCol = 0;
    int maxCol = cols / 2 - 1;
    if (dst_x >= cols / 2)
    {
        minCol = cols / 2;
        maxCol = cols - 1;
    }

    if (dst_y < rows && dst_x < cols)
    {
        float3 sum = make_float3(0.0f);

        for (int y = -kernelSizeDiv2; y <= kernelSizeDiv2; y++)
        {
            for (int x = -kernelSizeDiv2; x <= kernelSizeDiv2; x++)
            {
                int src_x = clamp(x + dst_x, minCol, maxCol);

                uchar3 pixel = src(src_y, src_x);
                float weight = kernelMat(y + kernelSizeDiv2, x + kernelSizeDiv2);

                sum.x += pixel.x * weight;
                sum.y += pixel.y * weight;
                sum.z += pixel.z * weight;
            }
        }

        dst(dst_y, dst_x).x = min(max(int(sum.x), 0), 255);
        dst(dst_y, dst_x).y = min(max(int(sum.y), 0), 255);
        dst(dst_y, dst_x).z = min(max(int(sum.z), 0), 255);
    }
}

inline int divUp(int a, int b)
{
    return ((a % b) != 0) ? (a / b + 1) : (a / b);
}

void processGaussianCUDA(
    const cv::cuda::GpuMat &src,
    cv::cuda::GpuMat &dst,
    const int kernelSizeDiv2,
    const cv::cuda::GpuMat &kernelMat)
{

    const dim3 block(32, 8);
    const dim3 grid(divUp(dst.cols, block.x), divUp(dst.rows, block.y));

    applyGaussianFilter<<<grid, block>>>(
        cv::cuda::PtrStep<uchar3>(src),
        cv::cuda::PtrStep<uchar3>(dst),
        dst.rows,
        dst.cols,
        kernelSizeDiv2,
        cv::cuda::PtrStep<float>(kernelMat));
}

void processAnaglyphCUDA(cv::cuda::GpuMat &src, cv::cuda::GpuMat &dst, const AnaglyphFuncion &selectedAnaglyph)
{
    const dim3 block(32, 8);
    const dim3 grid(divUp(dst.cols, block.x), divUp(dst.rows, block.y));

    selectedAnaglyph<<<grid, block>>>(src, dst, dst.rows, dst.cols);
}

// ------------------------------

int main(int argc, char **argv)
{

    if (argc < 5)
    {
        cout << "Usage: " << argv[0] << " <image> <anaglyphType> <kernelSizeDiv2> <sigma>" << endl;
        cout << "anaglyphType: true, gray, color, halfColor, optimized" << endl;
        return -1;
    }

    // parse arguments
    const char *filename = argv[1];
    const char *anaglyphType = argv[2];
    const int kernelSizeDiv2 = atoi(argv[3]);
    const float sigma = atof(argv[4]);

    cout << "   Filename: " << filename << endl;
    cout << "   Anaglyph: " << anaglyphType << endl;
    cout << "Kernel size: " << kernelSizeDiv2 << endl;
    cout << "      Sigma: " << sigma << endl;

    const AnaglyphFuncion selectedAnaglyph = selectAnaglyphFunction(anaglyphType);

    if (selectedAnaglyph == nullptr)
    {
        cout << "Invalid anaglyph type: " << anaglyphType << endl;
        cout << "anaglyphType: true, gray, color, halfColor, optimized" << endl;
        return -1;
    }

    const cv::Mat h_src = cv::imread(filename, cv::IMREAD_COLOR);
    cv::Mat h_dst;
    h_dst.create(h_src.rows, h_src.cols / 2, CV_8UC3);

    // gaussian kernel
    cv::Mat_<float> k_kernelMat(2 * kernelSizeDiv2 + 1, 2 * kernelSizeDiv2 + 1);
    makeGaussianKernel(kernelSizeDiv2, sigma, k_kernelMat);

    // upload kernel to GPU
    cv::cuda::GpuMat d_kernelMat;
    d_kernelMat.upload(k_kernelMat);

    cv::cuda::GpuMat d_src, d_mid, d_dst;

    auto begin = chrono::high_resolution_clock::now();
    const int iter = 100;

    for (int i = 0; i < iter; i++)
    {
        // upload source and destination images
        d_src.upload(h_src);
        d_mid.upload(h_src);
        d_dst.upload(h_dst);
        // process
        processGaussianCUDA(d_src, d_mid, kernelSizeDiv2, d_kernelMat);
        processAnaglyphCUDA(d_mid, d_dst, selectedAnaglyph);
        // download destination image
        d_dst.download(h_dst);
    }
    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> diff = end - begin;

    cout << "Time: " << diff.count() << endl;
    cout << "Time/frame: " << diff.count() / iter << endl;
    cout << "IPS: " << iter / diff.count() << endl;

    cv::imwrite("./results/original.png", h_src);
    cv::imwrite("./results/gaussian.png", h_dst);

    return 0;
}
